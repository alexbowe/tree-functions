#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <limits>
#include <cassert>
//#include <thrust/functional.h>
//#include "PseudoTree.h"
#include <string>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform_scan.h>
#include <thrust/tuple.h>
#include <thrust/device_vector.h>

#include "debug.h"

struct map_parenth : public thrust::unary_function<char,int>
{
    // Maps ( to 1 and ) to 0
    __device__ __host__
    int operator()(char p) const
    {
        return (p - '(') ^ 1;
    }
};

//struct Tree
// Just has two device vectors, maybe more when I support bitmaps...
// provide access using a zip iterator
typedef int block_t;
static const unsigned int block_bytes = sizeof(block_t);
// How many bits? Add 1 if it is signed (digits returns magnitude-bits only).
static const unsigned int block_bits  = std::numeric_limits<block_t>::digits
    + (std::numeric_limits<block_t>::is_signed? 1 : 0);

typedef int value_t;
typedef thrust::tuple<int, int, int, int, value_t, value_t, value_t> ptree;
typedef thrust::plus<value_t> plus;
typedef thrust::maximum<value_t> maximum;
typedef thrust::negate<value_t> negate;
const int plus_id = 0;
const int maximum_id = INT_MIN;

template <value_t id1, value_t id2>
struct make_initial_ptree : public thrust::unary_function<thrust::tuple<block_t,value_t>,ptree>
{
    // ( -> (1,   0,  1,  1, w,   w, id2)
    // ) -> (-1, -1, -1, -1, w, id1, id2)
    template <typename Tuple>
    __host__ __device__
    ptree operator()(Tuple t)
    {
        int p = thrust::get<0>(t);
        int w = thrust::get<1>(t);
        return p? ptree(1, 0, 1, 1, w, w, id2) : ptree(-1, -1, -1, -1, w, id1, id2);
    }
};

template <typename Op1, typename Op2, typename Inv>
struct merge_ptrees : public thrust::binary_function<ptree, ptree, ptree>
{
    __host__ __device__
    ptree operator()(const ptree& a, const ptree& b)
    {
        // Let's give these tuple members nicer names (that coincide with the paper)
        int L1 = thrust::get<0>(a); int L2 = thrust::get<0>(b);
        int M1 = thrust::get<1>(a); int M2 = thrust::get<1>(b);
        int B1 = thrust::get<2>(a); int B2 = thrust::get<2>(b);
        int E1 = thrust::get<3>(a); int E2 = thrust::get<3>(b);
        value_t R1 = thrust::get<4>(a); value_t R2 = thrust::get<4>(b);
        value_t A1 = thrust::get<5>(a); value_t A2 = thrust::get<5>(b);
        value_t F1 = thrust::get<6>(a); value_t F2 = thrust::get<6>(b);

        int L = L1 + L2;
        int M = min(M1, L1 + M2);
        int B = B1? B1 : B2;
        int E = E2? E2 : E1;
        value_t R = op1(R1, R2);
        value_t A = (M == M1)? op1(A1, R2) : A2;
        value_t F_prime = (E1 == 1 && B2 == -1)? op2(F1, A1) : F1;
        value_t H = op1(op1(A1, R2), inv(A2));
        value_t F = (M == M1)? op2(F_prime, op1(H, F2)) : op2(op1(inv(H), F_prime), F2);
        
        return ptree(L, M, B, E, R, A, F);
    }
     
    static Op1 op1;
    static Op2 op2;
    static Inv inv;
};


template <typename Op1, typename Op2, typename Inv>
Op1 merge_ptrees<Op1, Op2, Inv>::op1 = Op1();

template <typename Op1, typename Op2, typename Inv>
Op2 merge_ptrees<Op1, Op2, Inv>::op2 = Op2();

template <typename Op1, typename Op2, typename Inv>
Inv merge_ptrees<Op1, Op2, Inv>::inv = Inv();

int main(void)
{
    std::string s = "(()((()())())(()())(()))";
    thrust::host_vector<block_t> temp(s.begin(), s.end());
    // Later, allocate blocks of bits
    // unsigned int num_blocks = (s.length() + block_bits - 1) / block_bits;
    thrust::device_vector<block_t> p(thrust::make_transform_iterator(temp.begin(), map_parenth()),
                                     thrust::make_transform_iterator(temp.end(),   map_parenth()));

    // Write weight matching function that accepts iterators for chars, iterators for weights
    // input parenths, input level-order weights, output weight array
    value_t weights[] = {2, 3, -3, -1, 6, 4, -4, 9, -9, -6, 5, -5, 1, 1, -2, 2, 6, -6, -1, 4, 7, -7, -5, -2};
    thrust::device_vector<value_t> v(weights, weights + s.length());
    
    TRACE_V((p));
    TRACE_V((v));

    // allocate space for num_blocks pseudotrees
    thrust::device_vector<int>     L_v(s.size());
    thrust::device_vector<int>     M_v(s.size());
    thrust::device_vector<int>     B_v(s.size());
    thrust::device_vector<int>     E_v(s.size());
    thrust::device_vector<value_t> R_v(s.size());
    thrust::device_vector<value_t> A_v(s.size());
    thrust::device_vector<value_t> F_v(s.size());

    //thrust::make_transform_iterator(
    thrust::transform_inclusive_scan(
        thrust::make_zip_iterator(thrust::make_tuple(p.begin(), v.begin())),
        thrust::make_zip_iterator(thrust::make_tuple(p.end(), v.end())),
        thrust::make_zip_iterator(thrust::make_tuple(L_v.begin(), M_v.begin(), B_v.begin(), E_v.begin(), R_v.begin(),
            A_v.begin(), F_v.begin())),
        make_initial_ptree<plus_id, maximum_id>(),
        merge_ptrees<plus, maximum, negate>()
    );

    TRACE_V((L_v));
    TRACE_V((M_v));
    TRACE_V((B_v));
    TRACE_V((E_v));
    TRACE_V((R_v));
    TRACE_V((A_v));
    TRACE_V((F_v));

    return 0;
}
